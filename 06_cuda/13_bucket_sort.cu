
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <vector>


__global__ void countElements(int *input, int *counts, int n, int range) {
    
    extern __shared__ int sharedCounts[];
    
    for (int i = threadIdx.x; i < range; i += blockDim.x) {
        sharedCounts[i] = 0;
    }
    __syncthreads();
    
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
        atomicAdd(&sharedCounts[input[i]], 1);
    }
    __syncthreads();
    
    for (int i = threadIdx.x; i < range; i += blockDim.x) {
        atomicAdd(&counts[i], sharedCounts[i]);
    }
}

__global__ void sortElements(int *input, int *output, int *prefixSums, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    int value = input[i];
    // Use atomicAdd on global prefixSums to get unique position
    int pos = atomicAdd(&prefixSums[value], 1);
    output[pos] = value;
}

int main() {
    int n = 50;
    int range = 5;
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    std::vector<int> key(n);
    
    printf("Unsorted array: ");
    srand(time(0));
    for (int i = 0; i < n; i++) {
        key[i] = rand() % range;
        printf("%d ", key[i]);
    }
    printf("\n");
    
    int *d_input, *d_output, *d_counts;
    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_counts, range * sizeof(int));
    
    hipMemset(d_counts, 0, range * sizeof(int));
    
    hipMemcpy(d_input, key.data(), n * sizeof(int), hipMemcpyHostToDevice);
    
    countElements<<<numBlocks, blockSize, range * sizeof(int)>>>(d_input, d_counts, n, range);
    hipDeviceSynchronize();
    std::vector<int> h_counts(range);
    hipMemcpy(h_counts.data(), d_counts, range * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i = 0; i < range; i++) {
        int temp = h_counts[i];
        h_counts[i] = sum;
        sum += temp;
    }
    hipMemcpy(d_counts, h_counts.data(), range * sizeof(int), hipMemcpyHostToDevice);
      
    sortElements<<<numBlocks, blockSize>>>(d_input, d_output, d_counts, n);
    
    hipDeviceSynchronize();
    
    std::vector<int> output(n);
    hipMemcpy(output.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Sorted array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");
    
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_counts);
    
    return 0;
}