#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;
typedef vector<vector<double>> matrix;

__global__ void computeBKernel(double* u, double* v, double* b, int nx, int ny, 
                              double dx, double dy, double dt, double rho) {

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i < nx - 1 && j < ny - 1) {
        double du_dx = (u[j*nx + i+1] - u[j*nx + i-1]) / (2.0f * dx);
        double dv_dy = (v[(j+1)*nx + i] - v[(j-1)*nx + i]) / (2.0f * dy);

        double du_dy = (u[(j+1)*nx + i] - u[(j-1)*nx + i]) / (2.0f * dy);
        double dv_dx = (v[j*nx + i+1] - v[j*nx + i-1]) / (2.0f * dx);

        double div_term = (du_dx + dv_dy) / dt;
        double squared_term = du_dy*du_dy + dv_dx*dv_dx;
        double cross_term = 2.0f * du_dy * dv_dx;

        b[j*nx + i] = rho * (div_term - squared_term - cross_term);

    }
}

__global__ void pressureIterationKernel(double* p, double* pn, double* b, int nx, int ny, 
                                       double dx, double dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i < nx && j < ny) {
        pn[j * nx + i] = p[j * nx + i];
    }

    __syncthreads();

    if (i > 0 && j > 0 && i < nx - 1 && j < ny - 1) {
        double pressure_x_term = dy*dy * (pn[j * nx + i + 1] + pn[j * nx + i - 1]);
        double pressure_y_term = dx*dx * (pn[(j + 1) * nx + i] + pn[(j - 1) * nx + i]);
        double denominator = 2.0 * (dx*dx + dy*dy);
        double source_term = b[j * nx + i] * dx*dx * dy*dy;

        p[j * nx + i] = (pressure_x_term + pressure_y_term - source_term) / denominator;
    }
    __syncthreads();
}

__global__ void pressureBoundaryKernel(double* p, int nx, int ny) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < ny) {
        p[j * nx] = p[j * nx + 1];
        p[j * nx + nx - 1] = p[j * nx + nx - 2];
    }
    __syncthreads();

    if (i < nx) {
        p[i] = p[nx + i];
        p[(ny - 1) * nx + i] = 0;
    }
    __syncthreads();
}

__global__ void velocityKernel(double* u, double* v, double* un, double* vn, double* p,
                              int nx, int ny, double dx, double dy, double dt, double rho, double nu) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (i < nx && j < ny) {
        un[j * nx + i] = u[j * nx + i];
        vn[j * nx + i] = v[j * nx + i];
    }
    __syncthreads();

    if (i > 0 && j > 0 && i < nx - 1 && j < ny - 1) {
        double u_prev = un[j * nx + i];    
        double u_conv_x = un[j * nx + i] * dt / dx * (un[j * nx + i] - un[j * nx + i - 1]);
        double u_conv_y = vn[j * nx + i] * dt / dy * (un[j * nx + i] - un[(j - 1) * nx + i]);
        double u_pressure = dt / (2 * rho * dx) * (p[j * nx + i + 1] - p[j * nx + i - 1]);
        double u_diff_x = nu * dt / (dx * dx) * (un[j * nx + i + 1] - 2 * un[j * nx + i] + un[j * nx + i - 1]);
        double u_diff_y = nu * dt / (dy * dy) * (un[(j + 1) * nx + i] - 2 * un[j * nx + i] + un[(j - 1) * nx + i]);
        
        u[j * nx + i] = u_prev - u_conv_x - u_conv_y - u_pressure + u_diff_x + u_diff_y;

        double v_prev = vn[j * nx + i];
        double v_conv_x = un[j * nx + i] * dt / dx * (vn[j * nx + i] - vn[j * nx + i - 1]);
        double v_conv_y = vn[j * nx + i] * dt / dy * (vn[j * nx + i] - vn[(j - 1) * nx + i]);
        double v_pressure = dt / (2 * rho * dy) * (p[(j + 1) * nx + i] - p[(j - 1) * nx + i]);
        double v_diff_x = nu * dt / (dx * dx) * (vn[j * nx + i + 1] - 2 * vn[j * nx + i] + vn[j * nx + i - 1]);
        double v_diff_y = nu * dt / (dy * dy) * (vn[(j + 1) * nx + i] - 2 * vn[j * nx + i] + vn[(j - 1) * nx + i]);

        v[j * nx + i] = v_prev - v_conv_x - v_conv_y - v_pressure + v_diff_x + v_diff_y;
    }
    __syncthreads();
}

__global__ void velocityBoundaryKernel(double* u, double* v, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < ny) {
        u[idx * nx] = 0;
        u[idx * nx + nx - 1] = 0;
        v[idx * nx] = 0;
        v[idx * nx + nx - 1] = 0;
    }

    if (idx < nx) {
        u[idx] = 0;
        u[(ny - 1) * nx + idx] = 1;
        v[idx] = 0;
        v[(ny - 1) * nx + idx] = 0;
    }
    __syncthreads();
}

#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)
void check_cuda_error(hipError_t result, const char* func, const char* file, int line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}

void copyMatrixToDevice(const matrix& host_matrix, double* device_array, int nx, int ny) {
    double* temp_data = new double[nx * ny];
    
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            temp_data[j * nx + i] = host_matrix[j][i];
        }
    }
    
    CHECK_CUDA_ERROR(hipMemcpy(device_array, temp_data, nx * ny * sizeof(double), 
                               hipMemcpyHostToDevice));
    
    delete[] temp_data;
}

void copyDeviceToMatrix(double* device_array, matrix& host_matrix, int nx, int ny) {
    double* temp_data = new double[nx * ny];
    
    CHECK_CUDA_ERROR(hipMemcpy(temp_data, device_array, nx * ny * sizeof(double), 
                               hipMemcpyDeviceToHost));
    
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            host_matrix[j][i] = temp_data[j * nx + i];
        }
    }
    
    delete[] temp_data;
}

int main() {
    int nx = 41;
    int ny = 41;
    int nt = 500;
    int nit = 50;
    double dx = 2.0f / (nx - 1);
    double dy = 2.0f / (ny - 1);
    double dt = 0.01f;
    double rho = 1.0f;
    double nu = 0.02f;
    
    matrix u(ny, vector<double>(nx, 0.0f));
    matrix v(ny, vector<double>(nx, 0.0f));
    matrix p(ny, vector<double>(nx, 0.0f));
    matrix b(ny, vector<double>(nx, 0.0f));
    matrix un(ny, vector<double>(nx, 0.0f));
    matrix vn(ny, vector<double>(nx, 0.0f));
    matrix pn(ny, vector<double>(nx, 0.0f));
    
    double *d_u, *d_v, *d_p, *d_b, *d_un, *d_vn, *d_pn;
    size_t size = nx * ny * sizeof(double);

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_u, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_v, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_p, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_un, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_vn, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_pn, size));

    CHECK_CUDA_ERROR(hipMemset(d_u, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_v, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_p, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_b, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_un, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_vn, 0, size));
    CHECK_CUDA_ERROR(hipMemset(d_pn, 0, size));

    copyMatrixToDevice(u, d_u, nx, ny);
    copyMatrixToDevice(v, d_v, nx, ny);
    copyMatrixToDevice(p, d_p, nx, ny);

    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    int maxBoundarySize = max(nx, ny);
    dim3 boundaryBlockSize(256);
    dim3 boundaryGridSize((maxBoundarySize + boundaryBlockSize.x - 1) / boundaryBlockSize.x);

    ofstream ufile("u.dat");
    ofstream vfile("v.dat");
    ofstream pfile("p.dat");
    
    for (int n = 0; n < nt; n++) {
        copyMatrixToDevice(u, d_u, nx, ny);
        copyMatrixToDevice(v, d_v, nx, ny);
        copyMatrixToDevice(p, d_p, nx, ny);

        printf("Computing B matrix...\n");
        computeBKernel<<<grid, block>>>(d_u, d_v, d_b, nx, ny, dx, dy, dt, rho);
        CHECK_CUDA_ERROR(hipGetLastError());

        
        for (int it = 0; it < nit; it++) {
            printf("Pressure iteration %d...\n", it);
            pressureIterationKernel<<<grid, block>>>(d_p, d_pn, d_b, nx, ny, dx, dy);
            CHECK_CUDA_ERROR(hipGetLastError());
            
            printf("Pressure boundary conditions...\n");
            pressureBoundaryKernel<<<boundaryGridSize, boundaryBlockSize>>>(d_p, nx, ny);
            CHECK_CUDA_ERROR(hipGetLastError());
        }
        
        printf("Updating velocity...\n");
        velocityKernel<<<grid, block>>>(d_u, d_v, d_un, d_vn, d_p, nx, ny, dx, dy, dt, rho, nu);
        CHECK_CUDA_ERROR(hipGetLastError());
        
        printf("Applying velocity boundary conditions...\n");
        velocityBoundaryKernel<<<boundaryGridSize, boundaryBlockSize>>>(d_u, d_v, nx, ny);
        CHECK_CUDA_ERROR(hipGetLastError());
        
        printf("Copying results back to host...\n");
        if (n % 10 == 0) {
            copyDeviceToMatrix(d_u, u, nx, ny);
            copyDeviceToMatrix(d_v, v, nx, ny);
            copyDeviceToMatrix(d_p, p, nx, ny);
        
            for (int j = 0; j < ny; j++) {
                for (int i = 0; i < nx; i++) {
                    ufile << u[j][i] << " ";
                }
            }
            ufile << "\n";
            
            for (int j = 0; j < ny; j++) {
                for (int i = 0; i < nx; i++) {
                    vfile << v[j][i] << " ";
                }
            }
            vfile << "\n";
            
            for (int j = 0; j < ny; j++) {
                for (int i = 0; i < nx; i++) {
                    pfile << p[j][i] << " ";
                }
            }
            pfile << "\n";
            
            printf("Time step %d completed\n", n);
        }
    }
    
    // Close files
    ufile.close();
    vfile.close();
    pfile.close();
    
    // Free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_p);
    hipFree(d_b);
    hipFree(d_un);
    hipFree(d_vn);
    hipFree(d_pn);
    
    return 0;
}